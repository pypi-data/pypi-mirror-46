#include "hip/hip_runtime.h"
/*=========================================================================
 *
 *  Copyright RTK Consortium
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *=========================================================================*/

/*****************
*  rtk #includes *
*****************/
#include "rtkCudaUtilities.hcu"
#include "rtkConfiguration.h"
#include "rtkCudaIntersectBox.hcu"
#include "rtkCudaWarpForwardProjectionImageFilter.hcu"

/*****************
*  C   #includes *
*****************/
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cmath>

/*****************
* CUDA #includes *
*****************/
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

// TEXTURES AND CONSTANTS //
texture<float, 3, hipReadModeElementType> tex_xdvf;
texture<float, 3, hipReadModeElementType> tex_ydvf;
texture<float, 3, hipReadModeElementType> tex_zdvf;
texture<float, 3, hipReadModeElementType> tex_vol;

__constant__ int3 c_projSize;
__constant__ float3 c_boxMin;
__constant__ float3 c_boxMax;
__constant__ float3 c_spacing;
__constant__ int3 c_volSize;
__constant__ float c_tStep;
__constant__ float c_matrices[SLAB_SIZE * 12]; //Can process stacks of at most SLAB_SIZE projections
__constant__ float c_sourcePos[SLAB_SIZE * 3]; //Can process stacks of at most SLAB_SIZE projections

__constant__ float c_IndexInputToPPInputMatrix[12];
__constant__ float c_IndexInputToIndexDVFMatrix[12];
__constant__ float c_PPInputToIndexInputMatrix[12];

//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
// K E R N E L S -_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_( S T A R T )_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_

// KERNEL kernel_forwardProject
__global__
void kernel_warped_forwardProject(float *dev_proj_in, float *dev_proj_out)
{
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned int numThread = j*c_projSize.x + i;

  if (i >= c_projSize.x || j >= c_projSize.y)
    return;

  // Setting ray origin
  Ray ray;
  float3 pixelPos;
  float tnear, tfar;

  for (unsigned int proj = 0; proj<c_projSize.z; proj++)
    {
    // Setting ray origin
    ray.o = make_float3(c_sourcePos[3 * proj], c_sourcePos[3 * proj + 1], c_sourcePos[3 * proj + 2]);

    pixelPos = matrix_multiply(make_float3(i,j,0), &(c_matrices[12*proj]));

    ray.d = pixelPos - ray.o;
    ray.d = ray.d / sqrtf(dot(ray.d,ray.d));

    // Detect intersection with box
    if ( !intersectBox(ray, &tnear, &tfar, c_boxMin, c_boxMax) || tfar < 0.f )
      {
      dev_proj_out[numThread + proj * c_projSize.x * c_projSize.y] = dev_proj_in[numThread + proj * c_projSize.x * c_projSize.y];
      }
    else
      {
      if (tnear < 0.f)
        tnear = 0.f; // clamp to near plane

      // Step length in mm
      float3 dirInMM = c_spacing * ray.d;
      float vStep = c_tStep / sqrtf(dot(dirInMM, dirInMM));
      float3 step = vStep * ray.d;

      // First position in the box
      float halfVStep = 0.5f*vStep;
      tnear = tnear + halfVStep;
      float3 pos = ray.o + tnear*ray.d;

      float  t;
      float  sample = 0.0f;
      float  sum    = 0.0f;

      float3 IndexInDVF, Displacement, PP, IndexInInput;

      for(t=tnear; t<=tfar; t+=vStep)
        {
        IndexInDVF = matrix_multiply(pos, c_IndexInputToIndexDVFMatrix);

        // Get each component of the displacement vector by
        // interpolation in the dvf
        Displacement.x = tex3D(tex_xdvf, IndexInDVF.x + 0.5f, IndexInDVF.y + 0.5f, IndexInDVF.z + 0.5f);
        Displacement.y = tex3D(tex_ydvf, IndexInDVF.x + 0.5f, IndexInDVF.y + 0.5f, IndexInDVF.z + 0.5f);
        Displacement.z = tex3D(tex_zdvf, IndexInDVF.x + 0.5f, IndexInDVF.y + 0.5f, IndexInDVF.z + 0.5f);

        // Matrix multiply to get the physical coordinates of the current point in the output volume
        // + the displacement
        PP = matrix_multiply(pos, c_IndexInputToPPInputMatrix) + Displacement;

        // Convert it to a continuous index
        IndexInInput = matrix_multiply(PP, c_PPInputToIndexInputMatrix);

        // Read from 3D texture from volume
        sample = tex3D(tex_vol, IndexInInput.x, IndexInInput.y, IndexInInput.z);

        // Accumulate, and move forward along the ray
        sum += sample;
        pos += step;
        }
      dev_proj_out[numThread + proj * c_projSize.x * c_projSize.y] = dev_proj_in[numThread + proj * c_projSize.x * c_projSize.y] + (sum+(tfar-t+halfVStep)/vStep*sample) * c_tStep;
      }
    }
}

//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
// K E R N E L S -_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-( E N D )-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_

///////////////////////////////////////////////////////////////////////////
// FUNCTION: CUDA_forward_project() //////////////////////////////////
void
CUDA_warp_forward_project( int projSize[3],
                           int volSize[3],
                           int dvfSize[3],
                           float* matrices,
                           float *dev_proj_in,
                           float *dev_proj_out,
                           float *dev_vol,
                           float t_step,
                           float* source_positions,
                           float box_min[3],
                           float box_max[3],
                           float spacing[3],
                           float *dev_input_dvf,
                           float IndexInputToIndexDVFMatrix[12],
                           float PPInputToIndexInputMatrix[12],
                           float IndexInputToPPInputMatrix[12] )
{
  // Create CUBLAS context
  hipblasHandle_t  handle;
  hipblasCreate(&handle);

  // constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(c_projSize), projSize, sizeof(int3));
  hipMemcpyToSymbol(HIP_SYMBOL(c_boxMin), box_min, sizeof(float3));
  hipMemcpyToSymbol(HIP_SYMBOL(c_boxMax), box_max, sizeof(float3));
  hipMemcpyToSymbol(HIP_SYMBOL(c_spacing), spacing, sizeof(float3));
  hipMemcpyToSymbol(HIP_SYMBOL(c_volSize), volSize, sizeof(int3));
  hipMemcpyToSymbol(HIP_SYMBOL(c_tStep), &t_step, sizeof(float));

  // Copy the source position matrix into a float3 in constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(c_sourcePos), &(source_positions[0]), 3 * sizeof(float) * projSize[2]);

  // Copy the projection matrices into constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(c_matrices), &(matrices[0]), 12 * sizeof(float) * projSize[2]);

  // Prepare channel description for arrays
  static hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

  // Extent stuff, will be used for each component extraction
  hipExtent dvfExtent = make_hipExtent(dvfSize[0], dvfSize[1], dvfSize[2]);

  // Set texture parameters for the input volume
  tex_vol.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
  tex_vol.addressMode[1] = hipAddressModeClamp;
  tex_vol.addressMode[2] = hipAddressModeClamp;
  tex_vol.normalized = false;                     // access with normalized texture coordinates
  tex_vol.filterMode = hipFilterModeLinear;      // linear interpolation

  // Copy volume data to array, bind the array to the texture
  hipExtent volExtent = make_hipExtent(volSize[0], volSize[1], volSize[2]);
  hipArray *array_vol;
  hipMalloc3DArray((hipArray**)&array_vol, &channelDesc, volExtent);
  CUDA_CHECK_ERROR;

  // Copy data to 3D array
  hipMemcpy3DParms copyParams = hipMemcpy3DParms();
  copyParams.srcPtr   = make_hipPitchedPtr(dev_vol, volSize[0]*sizeof(float), volSize[0], volSize[1]);
  copyParams.dstArray = (hipArray*)array_vol;
  copyParams.extent   = volExtent;
  copyParams.kind     = hipMemcpyDeviceToDevice;
  hipMemcpy3D(&copyParams);
  CUDA_CHECK_ERROR;

  // Set texture parameters
  tex_xdvf.addressMode[0] = hipAddressModeBorder;
  tex_xdvf.addressMode[1] = hipAddressModeBorder;
  tex_xdvf.addressMode[2] = hipAddressModeBorder;
  tex_xdvf.filterMode = hipFilterModeLinear;
  tex_xdvf.normalized = false; // don't access with normalized texture coords

  tex_ydvf.addressMode[0] = hipAddressModeBorder;
  tex_ydvf.addressMode[1] = hipAddressModeBorder;
  tex_ydvf.addressMode[2] = hipAddressModeBorder;
  tex_ydvf.filterMode = hipFilterModeLinear;
  tex_ydvf.normalized = false;

  tex_zdvf.addressMode[0] = hipAddressModeBorder;
  tex_zdvf.addressMode[1] = hipAddressModeBorder;
  tex_zdvf.addressMode[2] = hipAddressModeBorder;
  tex_zdvf.filterMode = hipFilterModeLinear;
  tex_zdvf.normalized = false;

  // Allocate an intermediate memory space to extract x, y and z components of the DVF
  float *DVFcomponent;
  int numel = dvfSize[0] * dvfSize[1] * dvfSize[2];
  hipMalloc(&DVFcomponent, numel * sizeof(float));
  float one = 1.0;

  // Allocate the arrays used for textures
  hipArray** DVFcomponentArrays = new hipArray* [3];
  CUDA_CHECK_ERROR;

  // Copy image data to arrays. The tricky part is the make_hipPitchedPtr.
  // The best way to understand it is to read
  // http://stackoverflow.com/questions/16119943/how-and-when-should-i-use-pitched-pointer-with-the-cuda-api
  for (unsigned int component = 0; component < 3; component++)
    {
    // Reset the intermediate memory
    hipMemset((void *)DVFcomponent, 0, numel * sizeof(float));

    // Fill it with the current component
    float * pComponent = dev_input_dvf + component;
    hipblasSaxpy(handle, numel, &one, pComponent, 3, DVFcomponent, 1);

    // Allocate the hipArray and fill it with the current DVFcomponent
    hipMalloc3DArray((hipArray**)& DVFcomponentArrays[component], &channelDesc, dvfExtent);
    hipMemcpy3DParms CopyParams = hipMemcpy3DParms();
    CopyParams.srcPtr   = make_hipPitchedPtr(DVFcomponent, dvfSize[0] * sizeof(float), dvfSize[0], dvfSize[1]);
    CopyParams.dstArray = (hipArray*) DVFcomponentArrays[component];
    CopyParams.extent   = dvfExtent;
    CopyParams.kind     = hipMemcpyDeviceToDevice;
    hipMemcpy3D(&CopyParams);
    CUDA_CHECK_ERROR;
    }

  // Intermediate memory is no longer needed
  hipFree (DVFcomponent);

  // Bind 3D arrays to 3D textures
  hipBindTextureToArray(tex_xdvf, (hipArray*) DVFcomponentArrays[0], channelDesc);
  hipBindTextureToArray(tex_ydvf, (hipArray*) DVFcomponentArrays[1], channelDesc);
  hipBindTextureToArray(tex_zdvf, (hipArray*) DVFcomponentArrays[2], channelDesc);
  CUDA_CHECK_ERROR;

  // Copy matrices into constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(c_IndexInputToPPInputMatrix), IndexInputToPPInputMatrix, 12*sizeof(float), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(c_IndexInputToIndexDVFMatrix),  IndexInputToIndexDVFMatrix,  12*sizeof(float), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(c_PPInputToIndexInputMatrix),  PPInputToIndexInputMatrix,  12*sizeof(float), 0, hipMemcpyHostToDevice);

  ///////////////
  // RUN
  dim3 dimBlock  = dim3(16, 16, 1);
  dim3 dimGrid = dim3(iDivUp(projSize[0], dimBlock.x), iDivUp(projSize[1], dimBlock.y));

  // Bind 3D array to 3D texture
  hipBindTextureToArray(tex_vol, (hipArray*)array_vol, channelDesc);
  CUDA_CHECK_ERROR;

  kernel_warped_forwardProject <<< dimGrid, dimBlock >>> (dev_proj_in, dev_proj_out);

  hipUnbindTexture (tex_xdvf);
  hipUnbindTexture (tex_ydvf);
  hipUnbindTexture (tex_zdvf);
  hipUnbindTexture (tex_vol);
  CUDA_CHECK_ERROR;

  hipFreeArray ((hipArray*) DVFcomponentArrays[0]);
  hipFreeArray ((hipArray*) DVFcomponentArrays[1]);
  hipFreeArray ((hipArray*) DVFcomponentArrays[2]);
  delete[] DVFcomponentArrays;
  hipFreeArray ((hipArray*)array_vol);
  CUDA_CHECK_ERROR;

  // Destroy CUBLAS context
  hipblasDestroy(handle);
}
